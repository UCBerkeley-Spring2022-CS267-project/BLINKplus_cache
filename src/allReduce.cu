#include <cstdio>

#include <hip/hip_runtime.h>
#include <nccl.h>


#include "extern_lib.h"

ncclResult_t  ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
    // TODO: why it is printed twice?
    printf("My Implementation\n");
    return EXTERN_LIB(ncclAllReduce, sendbuff, recvbuff, count, datatype, op, comm, stream);
}    
